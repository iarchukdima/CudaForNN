#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 8192
#define OUTPUT_SIZE 10
#define TRAIN_SIZE 10000
#define TEST_SIZE 1000
#define BATCH_SIZE 32
#define EPOCHS 10
#define LEARNING_RATE 0.001

#define TILE_SIZE 16

typedef struct {
    float *weights1;
    float *weights2;
    float *bias1;
    float *bias2;
    float *grad_weights1;
    float *grad_weights2;
    float *grad_bias1;
    float *grad_bias2;
} NeuralNetwork;

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            hipDeviceReset(); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

void load_data(const char *filename, float *data, int size) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(1);
    }
    size_t read_size = fread(data, sizeof(float), size, file);
    if (read_size != size) {
        fprintf(stderr, "Error reading data: expected %d elements, got %zu\n", size, read_size);
        exit(1);
    }
    fclose(file);
}

void load_labels(const char *filename, int *labels, int size) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(1);
    }
    size_t read_size = fread(labels, sizeof(int), size, file);
    if (read_size != size) {
        fprintf(stderr, "Error reading labels: expected %d elements, got %zu\n", size, read_size);
        exit(1);
    }
    fclose(file);
}

void initialize_weights(float *weights, int size) {
    float scale = sqrtf(2.0f / size);
    for (int i = 0; i < size; i++) {
        weights[i] = ((float)rand() / RAND_MAX) * scale - (scale / 2.0f);
    }
}

void initialize_bias(float *bias, int size) {
    for (int i = 0; i < size; i++) {
        bias[i] = 0.0f;
    }
}

#define TILE_SIZE 16

__global__ void matmul_a_b_kernel(const float *A, const float *B, float *C, int m, int n, int k) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < m && t * TILE_SIZE + threadIdx.x < n) {
            tileA[threadIdx.y][threadIdx.x] = A[row * n + t * TILE_SIZE + threadIdx.x];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < k && t * TILE_SIZE + threadIdx.y < n) {
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * k + col];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < m && col < k) {
        C[row * k + col] = sum;
    }
}

#define TILE_SIZE 16

__global__ void matmul_a_bt_kernel(float *A, float *B, float *C, int m, int n, int k) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int a_col = t * TILE_SIZE + threadIdx.x;
        int b_col = t * TILE_SIZE + threadIdx.x;
        
        if (row < m && a_col < n) {
            As[threadIdx.y][threadIdx.x] = A[row * n + a_col];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if (col < k && b_col < n) {
            Bs[threadIdx.y][threadIdx.x] = B[col * n + b_col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += As[threadIdx.y][i] * Bs[threadIdx.x][i];
        }
        
        __syncthreads();
    }
    
    if (row < m && col < k) {
        C[row * k + col] = sum;
    }
}

#define TILE_SIZE 16

__global__ void matmul_at_b_kernel(float *A, float *B, float *C, int m, int n, int k) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float sum = 0.0f;
    
    for (int t = 0; t < (m + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int a_row = t * TILE_SIZE + threadIdx.y;
        int b_row = t * TILE_SIZE + threadIdx.y;
        
        if (a_row < m && row < n) {
            As[threadIdx.y][threadIdx.x] = A[a_row * n + row];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        if (b_row < m && col < k) {
            Bs[threadIdx.y][threadIdx.x] = B[b_row * k + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }
        
        __syncthreads();
        
        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += As[i][threadIdx.y] * Bs[i][threadIdx.x];
        }
        
        __syncthreads();
    }
    
    if (row < n && col < k) {
        C[row * k + col] = sum;
    }
}

__global__ void relu_kernel(float *x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] = fmaxf(0.0f, x[idx]);
    }
}

__global__ void bias_add_kernel(float *x, float *bias, int batch_size, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int b = idx / size;
    int i = idx % size;

    if (b < batch_size && i < size) {
        x[idx] += bias[i];
    }
}

__global__ void softmax_kernel(float *x, int batch_size, int size) {
    int b = blockIdx.x;
    int tid = threadIdx.x;
    
    if (b >= batch_size) return;
    
    extern __shared__ float shared[];
    float *max_shared = shared;
    float *sum_shared = &shared[blockDim.x];
    
    float max_val = -INFINITY;
    for (int i = tid; i < size; i += blockDim.x) {
        max_val = fmaxf(max_val, x[b * size + i]);
    }
    max_shared[tid] = max_val;
    __syncthreads();
    
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            max_shared[tid] = fmaxf(max_shared[tid], max_shared[tid + s]);
        }
        __syncthreads();
    }
    max_val = max_shared[0];
    __syncthreads();
    
    float sum = 0.0f;
    for (int i = tid; i < size; i += blockDim.x) {
        sum += expf(x[b * size + i] - max_val);
    }
    sum_shared[tid] = sum;
    __syncthreads();
    
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sum_shared[tid] += sum_shared[tid + s];
        }
        __syncthreads();
    }
    sum = sum_shared[0];
    __syncthreads();
    
    for (int i = tid; i < size; i += blockDim.x) {
        x[b * size + i] = fmaxf(expf(x[b * size + i] - max_val) / sum, 1e-7f);
    }
}

__global__ void clip_gradients_kernel(float *gradients, int size, float max_norm) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float grad = gradients[idx];
        if (grad > max_norm) {
            gradients[idx] = max_norm;
        } else if (grad < -max_norm) {
            gradients[idx] = -max_norm;
        }
    }
}

void forward(NeuralNetwork *nn, float *d_input, float *d_hidden, float *d_output, int batch_size) {
    dim3 block_size(32, 32);
    dim3 grid_size((HIDDEN_SIZE + block_size.x - 1) / block_size.x, (batch_size + block_size.y - 1) / block_size.y);

    matmul_a_b_kernel<<<grid_size, block_size, 2 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_input, nn->weights1, d_hidden, batch_size, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    bias_add_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, nn->bias1, batch_size, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    relu_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, batch_size * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    grid_size.x = (OUTPUT_SIZE + block_size.x - 1) / block_size.x;
    grid_size.y = (batch_size + block_size.y - 1) / block_size.y;
    matmul_a_b_kernel<<<grid_size, block_size, 2 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_hidden, nn->weights2, d_output, batch_size, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    bias_add_kernel<<<(batch_size * OUTPUT_SIZE + 255) / 256, 256>>>(d_output, nn->bias2, batch_size, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    softmax_kernel<<<batch_size, 256, 2 * 256 * sizeof(float) >>>(d_output, batch_size, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());
}

float cross_entropy_loss(float *output, int *labels, int batch_size) {
    float total_loss = 0.0f;
    for (int b = 0; b < batch_size; b++) {
        total_loss -= logf(fmaxf(output[b * OUTPUT_SIZE + labels[b]], 1e-7f));
    }
    return total_loss / batch_size;
}

__global__ void zero_grad_kernel(float *grad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad[idx] = 0.0f;
    }
}

__global__ void compute_output_gradients_kernel(float *grad_output, float *output, int *labels, int batch_size) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch_size) {
        for (int i = 0; i < OUTPUT_SIZE; ++i) {
            grad_output[b * OUTPUT_SIZE + i] = output[b * OUTPUT_SIZE + i];
        }
        grad_output[b * OUTPUT_SIZE + labels[b]] -= 1.0f;
    }
}

__global__ void update_gradients_kernel(float *grad_weights, float *grad_bias, float *grad_layer, float *prev_layer, int batch_size, int prev_size, int curr_size) {
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < curr_size && j < prev_size) {
        float grad_w_sum = 0.0f;
        for (int b = 0; b < batch_size; ++b) {
            grad_w_sum += grad_layer[b * curr_size + i] * prev_layer[b * prev_size + j];
        }
        atomicAdd(&grad_weights[i * prev_size + j], grad_w_sum);

        if (j == 0) {
            float grad_b_sum = 0.0f;
            for (int b = 0; b < batch_size; ++b) {
                grad_b_sum += grad_layer[b * curr_size + i];
            }
            atomicAdd(&grad_bias[i], grad_b_sum);
        }
    }
}

__global__ void drelu_kernel(float *x, float *d_ReLU_out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_ReLU_out[idx] = x[idx] > 0.0f ? 1.0f : 0.0f;
    }
}

__global__ void multiply_gradients_kernel(float *grad1, float *grad2, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad1[idx] *= grad2[idx];
    }
}

void backward(NeuralNetwork *nn, float *d_input, float *d_hidden, float *d_output, int *d_labels, int batch_size) {
    zero_grad_kernel<<<(HIDDEN_SIZE * INPUT_SIZE + 256 - 1) / 256, 256>>>(nn->grad_weights1, HIDDEN_SIZE * INPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    zero_grad_kernel<<<(OUTPUT_SIZE * HIDDEN_SIZE + 256 - 1) / 256, 256>>>(nn->grad_weights2, OUTPUT_SIZE * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    zero_grad_kernel<<<(HIDDEN_SIZE + 256 - 1) / 256, 256>>>(nn->grad_bias1, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    zero_grad_kernel<<<(OUTPUT_SIZE + 256 - 1) / 256, 256>>>(nn->grad_bias2, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    float *d_grad_output;
    CUDA_CHECK(hipMalloc(&d_grad_output, batch_size * OUTPUT_SIZE * sizeof(float)));
    compute_output_gradients_kernel<<<(batch_size + 255) / 256, 256>>>(d_grad_output, d_output, d_labels, batch_size);
    CUDA_CHECK(hipGetLastError());

    dim3 block_size(32, 32);
    dim3 grid_size((HIDDEN_SIZE + block_size.x - 1) / block_size.x, (OUTPUT_SIZE + block_size.y - 1) / block_size.y);
    matmul_at_b_kernel<<<grid_size, block_size, 2 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_hidden, d_grad_output, nn->grad_weights2, batch_size, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    update_gradients_kernel<<<grid_size, block_size>>>(nn->grad_weights2, nn->grad_bias2, d_grad_output, d_hidden, batch_size, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    float *d_dX2;
    CUDA_CHECK(hipMalloc(&d_dX2, batch_size * HIDDEN_SIZE * sizeof(float)));
    grid_size.x = (HIDDEN_SIZE + block_size.x - 1) / block_size.x;
    grid_size.y = (batch_size + block_size.y - 1) / block_size.y;
    matmul_a_bt_kernel<<<grid_size, block_size, 2 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_grad_output, nn->weights2, d_dX2, batch_size, OUTPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    float *d_grad_hidden;
    CUDA_CHECK(hipMalloc(&d_grad_hidden, batch_size * HIDDEN_SIZE * sizeof(float)));
    drelu_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, d_grad_hidden, batch_size * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    multiply_gradients_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_dX2, d_grad_hidden, batch_size * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    grid_size.x = (INPUT_SIZE + block_size.x - 1) / block_size.x;
    grid_size.y = (HIDDEN_SIZE + block_size.y - 1) / block_size.y;
    matmul_at_b_kernel<<<grid_size, block_size, 2 * TILE_SIZE * TILE_SIZE * sizeof(float)>>>(d_input, d_dX2, nn->grad_weights1, batch_size, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    update_gradients_kernel<<<grid_size, block_size>>>(nn->grad_weights1, nn->grad_bias1, d_dX2, d_input, batch_size, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    hipFree(d_grad_output);
    hipFree(d_dX2);
    hipFree(d_grad_hidden);
}
